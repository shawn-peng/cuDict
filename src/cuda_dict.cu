
#include "cuda_tuple.cuh"
typedef Tuple<uint32_t> Tuple1;
#include "cuda_dict.cuh"

__device__ extern uint32_t prime_factors[] = {31, 37, 41, 43, 47};
// static hipError_t cuda_ret = 0;

template
struct CUDA_Static_Dict<Tuple1, int32_t>;

template
std::ostream &operator << <Tuple1, int32_t>(std::ostream &os, const CUDA_Static_Dict<Tuple1, int32_t> &d);
